#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "../common/simpleTime.h"
#include "../common/CUDAErrorCheck.h"
#include "csr-utils.cuh"
#include "csr-UserFunctions.cuh"

#define JOB_FINISHED 1
#define JOB_NOT_FINISHED_YET 0

#define PHYSICAL_WARP_SIZE 32
#define COMPILE_TIME_DEFINED_BLOCK_SIZE 256

// Virtual Warp-Centric (VWC) manner of processing graph using Compressed Sparse Row (CSR) representation format.
// One can use shuffle instructions to implement this kernel. I used shared memory for data exchange between threads.
template < uint VWSize, uint VWMask >
__global__ void VWC_CSR_GPU_kernel(	const uint num_of_vertices,
									const uint* vertices_indices,
									const uint* edges_indices,
									Vertex* VertexValue,
									Edge* EdgeValue,
									Vertex_static* VertexValue_static,
									int* dev_finished) {

	__shared__ Vertex final_vertex_values[COMPILE_TIME_DEFINED_BLOCK_SIZE >> VWMask];
	__shared__ Vertex thread_outcome[COMPILE_TIME_DEFINED_BLOCK_SIZE];
	volatile __shared__ uint edges_starting_address[COMPILE_TIME_DEFINED_BLOCK_SIZE >> VWMask];
	volatile __shared__ uint ngbrs_size[COMPILE_TIME_DEFINED_BLOCK_SIZE >> VWMask];
	Vertex previous_vertex_value;

	// You might gain some performance if you limit maximum number of registers per thread with -maxrregcount flag. For example, specifying 32 for the Kepler architecture.
	const uint warp_in_block_offset = threadIdx.x >> VWMask;
	const uint VLane_id = threadIdx.x & (VWSize-1);
	const uint t_id = threadIdx.x + blockIdx.x * blockDim.x;
	const uint VW_id = t_id >> VWMask;
	if ( VW_id >= num_of_vertices )
		return;

	previous_vertex_value = VertexValue[VW_id];
	// Only one virtual lane in the virtual warp does vertex initialization.
	if ( VLane_id == 0 ) {
		edges_starting_address [ warp_in_block_offset ] = vertices_indices [ VW_id ];
		ngbrs_size [ warp_in_block_offset ] = vertices_indices [ VW_id + 1 ] - edges_starting_address [ warp_in_block_offset ] ;
		init_compute( final_vertex_values + warp_in_block_offset, &previous_vertex_value );
	}

	for ( uint index = VLane_id; index < ngbrs_size[ warp_in_block_offset ]; index += VWSize ) {

		uint target_edge = edges_starting_address[ warp_in_block_offset ] + index;
		uint target_vertex = edges_indices [ target_edge ];
		compute_local ( 	VertexValue + target_vertex,
							VertexValue_static + target_vertex,
							EdgeValue + target_edge,
							thread_outcome + threadIdx.x,
							&previous_vertex_value );

		// Parallel Reduction. Totally unrolled.
		if ( VWSize == 32 )
			if( VLane_id < 16 )
				if ( (index + 16) < ngbrs_size[ warp_in_block_offset ])
					compute_reduce ( thread_outcome + threadIdx.x, thread_outcome + threadIdx.x + 16 );
		if ( VWSize >= 16 )
			if( VLane_id < 8 )
				if ( (index + 8) < ngbrs_size[ warp_in_block_offset ])
					compute_reduce ( thread_outcome + threadIdx.x, thread_outcome + threadIdx.x + 8 );
		if ( VWSize >= 8 )
			if( VLane_id < 4 )
				if ( (index + 4) < ngbrs_size[ warp_in_block_offset ])
					compute_reduce ( thread_outcome + threadIdx.x, thread_outcome + threadIdx.x + 4 );
		if ( VWSize >= 4 )
			if( VLane_id < 2 )
				if ( (index + 2) < ngbrs_size[ warp_in_block_offset ])
					compute_reduce ( thread_outcome + threadIdx.x, thread_outcome + threadIdx.x + 2 );
		if ( VWSize >= 2 )
			if( VLane_id < 1 ) {
				if ( (index + 1) < ngbrs_size[ warp_in_block_offset ])
					compute_reduce ( thread_outcome + threadIdx.x, thread_outcome + threadIdx.x + 1 );
				compute_reduce ( final_vertex_values + warp_in_block_offset, thread_outcome + threadIdx.x );	//	Virtual lane 0 saves the final value of current iteration.
			}

	}

	if ( VLane_id == 0 )
		if ( update_condition ( final_vertex_values + warp_in_block_offset, &previous_vertex_value  ) ) {
			(*dev_finished) = JOB_NOT_FINISHED_YET;
			VertexValue [ VW_id ] = (Vertex) (final_vertex_values [ warp_in_block_offset ]);
		}

}


bool processGraphVWC(	CSRGraph* hostGraph,
						const uint VirtualWarpSize ) {

	// Variables collecting statistics info.
	float H2D_copy_time, processing_time, D2H_copy_time;

	// Getting current device properties to fully occupy SM available threads.
	int currentDevice;
	CUDAErrorCheck ( hipGetDevice( &currentDevice ) );
	hipDeviceProp_t deviceProp;
	CUDAErrorCheck ( hipGetDeviceProperties(&deviceProp, currentDevice) );

	dim3 blockDim( COMPILE_TIME_DEFINED_BLOCK_SIZE, 1, 1 );

	/*
	// Occupying all SMs with (constant) maximum number of threads to eliminate new block scheduling overhead.
	dim3 gridDim( (deviceProp.multiProcessorCount*deviceProp.maxThreadsPerMultiProcessor)/blockDim.x, 1, 1);
	 */

	dim3 gridDim( ceil(((double)hostGraph->num_of_vertices)/(COMPILE_TIME_DEFINED_BLOCK_SIZE/VirtualWarpSize)), 1, 1 );

	// Host and device flags indicating if the processing is finished.
	int finished;
	int* dev_finished;
	CUDAErrorCheck ( hipMalloc((void**)&dev_finished, sizeof(int)) );

	// Creation and memory allocation for the graph in device global memory.
	CSRGraph dev_Graph;
	dev_Graph.num_of_edges = hostGraph->num_of_edges;
	dev_Graph.num_of_vertices = hostGraph->num_of_vertices;
	CUDAErrorCheck ( hipMalloc((void**)&(dev_Graph.vertices_indices), ( dev_Graph.num_of_vertices + 1 ) * sizeof(unsigned int)) );
	CUDAErrorCheck ( hipMalloc((void**)&(dev_Graph.edges_indices), ( dev_Graph.num_of_edges ) * sizeof(unsigned int)) );
	CUDAErrorCheck ( hipMalloc((void**)&(dev_Graph.VertexValue), ( dev_Graph.num_of_vertices ) * sizeof(Vertex)) );
	if(sizeof(Edge)>0) CUDAErrorCheck ( hipMalloc((void**)&(dev_Graph.EdgeValue), ( dev_Graph.num_of_edges ) * sizeof(Edge)) );
	if(sizeof(Vertex_static)>0) CUDAErrorCheck ( hipMalloc((void**)&(dev_Graph.VertexValue_static), ( dev_Graph.num_of_vertices ) * sizeof(Vertex_static)) );

	// Copy the graph from the host to the device
	fprintf ( stdout, "Copying the graph from the host to the device ...\n" );
	setTime();

	CUDAErrorCheck ( hipMemcpyAsync( dev_Graph.vertices_indices, hostGraph->vertices_indices, ( dev_Graph.num_of_vertices + 1 ) * sizeof(unsigned int), hipMemcpyHostToDevice));
	CUDAErrorCheck ( hipMemcpyAsync( dev_Graph.edges_indices, hostGraph->edges_indices, ( dev_Graph.num_of_edges ) * sizeof(unsigned int), hipMemcpyHostToDevice));
	CUDAErrorCheck ( hipMemcpyAsync( dev_Graph.VertexValue, hostGraph->VertexValue, ( dev_Graph.num_of_vertices ) * sizeof(Vertex), hipMemcpyHostToDevice));
	if(sizeof(Edge)>0) CUDAErrorCheck ( hipMemcpyAsync( dev_Graph.EdgeValue, hostGraph->EdgeValue, ( dev_Graph.num_of_edges ) * sizeof(Edge), hipMemcpyHostToDevice));
	if(sizeof(Vertex_static)>0) CUDAErrorCheck ( hipMemcpyAsync( dev_Graph.VertexValue_static, hostGraph->VertexValue_static, ( dev_Graph.num_of_vertices ) * sizeof(Vertex_static), hipMemcpyHostToDevice));
	CUDAErrorCheck ( hipDeviceSynchronize() );

	H2D_copy_time = getTime();
	fprintf( stdout, "Copying the graph from the host to the device finished in: %f (ms)\n", H2D_copy_time );

	// Iteratively process the graph on the device.
	fprintf( stdout, "Processing graph in a virtual warp-centric manner using CSR representation ...\n" );
	unsigned int counter = 0;
	setTime();
	do {
		finished = JOB_FINISHED;
		CUDAErrorCheck ( hipMemcpyAsync ( dev_finished, &finished, sizeof(char), hipMemcpyHostToDevice ) );
		switch( VirtualWarpSize ) {
		case(32):
			VWC_CSR_GPU_kernel <32,5> <<< gridDim, blockDim >>> (	dev_Graph.num_of_vertices,
																	dev_Graph.vertices_indices,
																	dev_Graph.edges_indices,
																	dev_Graph.VertexValue,
																	dev_Graph.EdgeValue,
																	dev_Graph.VertexValue_static,
																	dev_finished );
			break;
		case(16):
			VWC_CSR_GPU_kernel <16,4> <<< gridDim, blockDim >>> (	dev_Graph.num_of_vertices,
																	dev_Graph.vertices_indices,
																	dev_Graph.edges_indices,
																	dev_Graph.VertexValue,
																	dev_Graph.EdgeValue,
																	dev_Graph.VertexValue_static,
																	dev_finished );
			break;
		case(8):
			VWC_CSR_GPU_kernel <8,3> <<< gridDim, blockDim >>> (	dev_Graph.num_of_vertices,
																	dev_Graph.vertices_indices,
																	dev_Graph.edges_indices,
																	dev_Graph.VertexValue,
																	dev_Graph.EdgeValue,
																	dev_Graph.VertexValue_static,
																	dev_finished );
			break;
		case(4):
			VWC_CSR_GPU_kernel <4,2> <<< gridDim, blockDim >>> (	dev_Graph.num_of_vertices,
																	dev_Graph.vertices_indices,
																	dev_Graph.edges_indices,
																	dev_Graph.VertexValue,
																	dev_Graph.EdgeValue,
																	dev_Graph.VertexValue_static,
																	dev_finished );
			break;
		case(2):
			VWC_CSR_GPU_kernel <2,1> <<< gridDim, blockDim >>> (	dev_Graph.num_of_vertices,
																	dev_Graph.vertices_indices,
																	dev_Graph.edges_indices,
																	dev_Graph.VertexValue,
																	dev_Graph.EdgeValue,
																	dev_Graph.VertexValue_static,
																	dev_finished );
			break;

		}

		CUDAErrorCheck ( hipPeekAtLastError() );
		CUDAErrorCheck ( hipMemcpy ( &finished, dev_finished, sizeof(char), hipMemcpyDeviceToHost ) );
		counter++;
	} while ( finished == JOB_NOT_FINISHED_YET );
	processing_time = getTime();
	fprintf( stdout, "Processing finished in: %f (ms)\n", processing_time);
	fprintf( stdout, "Performed %u iterations in total.\n", counter);

	// Copy resulted vertex values back from the device to the host.
	fprintf( stdout, "Copying final vertex values from the device to the host ...\n" );
	setTime();
	CUDAErrorCheck ( hipMemcpy( hostGraph->VertexValue, dev_Graph.VertexValue, ( dev_Graph.num_of_vertices ) * sizeof(Vertex), hipMemcpyDeviceToHost));
	D2H_copy_time = getTime();
	fprintf( stdout, "Copying final vertex values back from the device to the host finished in: %f (ms)\n", D2H_copy_time);
	fprintf( stdout, "Total Execution time was: %f (ms)\n", H2D_copy_time+processing_time+D2H_copy_time );

	// Free up allocated device memory.
	CUDAErrorCheck ( hipFree( dev_finished ) );
	CUDAErrorCheck ( hipFree( dev_Graph.vertices_indices ) );
	CUDAErrorCheck ( hipFree( dev_Graph.edges_indices ) );
	CUDAErrorCheck ( hipFree( dev_Graph.VertexValue ) );
	if(sizeof(Edge)>0) CUDAErrorCheck ( hipFree( dev_Graph.EdgeValue ) );
	if(sizeof(Vertex_static)>0) CUDAErrorCheck ( hipFree( dev_Graph.VertexValue_static ) );

	return(EXIT_SUCCESS);
}

void ExecuteVWC(	FILE* inputFile,
					const int suggestedVirtualWarpSize,
					FILE* outputFile,
					const int arbparam) {

	CSRGraph HostGraph;
	primitiveVertex* primitiveVertices = (primitiveVertex*) malloc ( sizeof(primitiveVertex));
	assert( primitiveVertices );
	init_primitiveVertexCSR ( primitiveVertices );
	fprintf( stdout, "Populating the graph ...\n" );
	populatePrimitiveVertices( &HostGraph, &primitiveVertices, inputFile, arbparam);

	if(HostGraph.num_of_edges == 0) {
		fprintf( stderr, "No edge could be read from the file. Make sure provided formatting matches defined specification.\n");
		exit(EXIT_FAILURE);
	}
	else {
		fprintf( stdout, "Graph is populated with %u vertices and %u edges.\n", HostGraph.num_of_vertices, HostGraph.num_of_edges );
	}

	// Allocate using page-locked host memory. Graph size is limited by the size of system DRAM.
	CUDAErrorCheck ( hipHostAlloc( (void**) &(HostGraph.vertices_indices), (HostGraph.num_of_vertices+1) * sizeof(unsigned int), hipHostMallocDefault ) );
	CUDAErrorCheck ( hipHostAlloc( (void**) &(HostGraph.edges_indices), HostGraph.num_of_edges * sizeof(unsigned int), hipHostMallocDefault ) );
	CUDAErrorCheck ( hipHostAlloc( (void**) &(HostGraph.VertexValue), HostGraph.num_of_vertices * sizeof(Vertex), hipHostMallocDefault ) );
	if(sizeof(Edge)>0) CUDAErrorCheck ( hipHostAlloc( (void**) &(HostGraph.EdgeValue), HostGraph.num_of_edges * sizeof(Edge), hipHostMallocDefault ) );
	if(sizeof(Vertex_static)>0) CUDAErrorCheck ( hipHostAlloc( (void**) &(HostGraph.VertexValue_static), HostGraph.num_of_vertices * sizeof(Vertex_static), hipHostMallocDefault ) );

	copyPrimitiveVerticesIntoCSRGraph(&HostGraph, &primitiveVertices);

	// Free up primitive vertices.
	for( int i = 0; i < HostGraph.num_of_vertices; ++i )
		delete_primitiveVertexCSR( primitiveVertices+i );
	free( primitiveVertices );

	unsigned int VirtualWarpSize = PHYSICAL_WARP_SIZE;

	if( (suggestedVirtualWarpSize % 2 == 0) && (suggestedVirtualWarpSize != 0) && suggestedVirtualWarpSize <= PHYSICAL_WARP_SIZE ) {
		VirtualWarpSize = suggestedVirtualWarpSize;
		fprintf( stdout, "Chosen virtual warp size: %u\n", VirtualWarpSize);
	}
	else
		fprintf( stdout, "Chosen virtual warp size is not 2, 4, 8, 16, or 32. Default virtual warp size %u is used.\n", VirtualWarpSize);

	if ( processGraphVWC(&HostGraph, VirtualWarpSize) == EXIT_FAILURE) {
		fprintf(stderr, "An error happened. Exiting.\n");
		exit(EXIT_FAILURE);
	}

	for ( int i = 0; i < HostGraph.num_of_vertices; ++i )
		printVertexOutput(i, HostGraph.VertexValue[i], outputFile);

	CUDAErrorCheck ( hipHostFree(HostGraph.vertices_indices) );
	CUDAErrorCheck ( hipHostFree(HostGraph.edges_indices) );
	CUDAErrorCheck ( hipHostFree(HostGraph.VertexValue) );
	if(sizeof(Edge)>0) CUDAErrorCheck ( hipHostFree(HostGraph.EdgeValue) );
	if(sizeof(Vertex_static)>0) CUDAErrorCheck ( hipHostFree(HostGraph.VertexValue_static) );

}
